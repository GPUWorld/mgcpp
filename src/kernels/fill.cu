#include "hip/hip_runtime.h"

//          Copyright RedPortal 2017 - 2017.
// Distributed under the Boost Software License, Version 1.0.
//    (See accompanying file LICENSE or copy at
//          http://www.boost.org/LICENSE_1_0.txt)

#include <mgcpp/kernels/bits/fill.cuh>
#include <cmath>

#define BLK 64

namespace mgcpp
{
    __global__  void
    mgblas_Sfill_impl(float* arr, float value, size_t n)
    {
	int const id = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ float shared[64];

	if(id >= n)
	    return;

	shared[threadIdx.x] = value;
	__syncthreads();

	arr[id] = shared[threadIdx.x];
    }

    __global__  void
    mgblas_Dfill_impl(double* arr, double value, size_t n)
    {
	int const id = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ double shared[64];

	if(id >= n)
	    return;

	shared[threadIdx.x] = value;
	__syncthreads();

	arr[id] = shared[threadIdx.x];
    }

    __global__  void
    mgblas_Cfill_impl(hipComplex* arr, hipComplex value, size_t n)
    {
        int const id = blockIdx.x * blockDim.x + threadIdx.x;
        __shared__ hipComplex shared[64];

        if(id >= n)
        return;

        shared[threadIdx.x] = value;
        __syncthreads();

        arr[id] = shared[threadIdx.x];
    }

    __global__  void
    mgblas_Zfill_impl(hipDoubleComplex* arr, hipDoubleComplex value, size_t n)
    {
        int const id = blockIdx.x * blockDim.x + threadIdx.x;
        __shared__ hipDoubleComplex shared[64];

        if(id >= n)
        return;

        shared[threadIdx.x] = value;
        __syncthreads();

        arr[id] = shared[threadIdx.x];
    }

     }
}
