#include <type_traits>

#include <hip/hip_runtime.h>

#include <mgcpp/cuda/initialize.hpp>

namespace mg
{
    template<typename ElemType,
	typename = std::enable_if<std::is_arithmetic<ElemType>::value>>
    ElemType* cuda_malloc(size_t size)
    {
	ElemType* ptr = nullptr;
	hipMalloc((void**)(&ptr), size * sizeof(ElemType));

	// if(!ptr)
	// {
	//     throw;
	// }
    }
}